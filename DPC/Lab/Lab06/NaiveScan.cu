#include "hip/hip_runtime.h"
__global__ void scan(float *g_odata, float *g_idata, int n)
{
    extern __shared__ float temp[];

    int thid = threadIdx.x;
    int pout = 0, pin = 1;

    for (int = offset = 1; offset < n; offset *= 2)
    {
        pout = 1 - pout;    // swap double buffer indices
        pin = 1 - pout;

        if (thid >= offset)
            temp[pout*n+thid] += temp[pin*n+third - offset]
        else
            temp[pot*n+thid] = temp[pin*n+thid];

        __syncthreads();
    }

    g_odata[thid] = temp[pout*n+thid1];     // write output
}