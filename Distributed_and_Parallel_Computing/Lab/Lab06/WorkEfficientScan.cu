#include "hip/hip_runtime.h"
// Name: Zhangda Xu
// Student ID: 2088192
//
// Assignment goals achieved:
//   - block scan
//   - full scan for large vectors
//   - bank conflict avoidance optimization
//
// Time to execute the different scans on a vector of 10,000,000 entries:
//   - Block scan without BCAO: *TIME-1*
//   - Block scan with BCAO: *TIME-2*
//   - Full scan without BCAO: *TIME-3*
//   - Full scan with BCAO: *TIME-4*
//
// CPU model:
// GPU model:
//
// Description
// A short description of any implementation details or performance improvement strategies that you successfully
// implemented and which improve upon a base level implementation of the target goals.

__global__ void prescan(float *g_odata, float *g_idata, int n)
{
    extern __shared__ float temp[]; // allocated on invocation

    int thid = threadIndx.x;
    int offset = 1;

    temp[2*thid] = g_idata[2*thid];
    temp[2*thid+1] = g_idata[2*thid+1];

    for (int d = n>>1; d > 0; d >>= 1)  // build sum in place up the tree
    {
        __syncthreads();

        if (thid < d)
        {
            int ai = offset*(2*thid+1)-1;
            int bi = offset*(2*thid+2)-1;

            temp[bi] += temp[ai];
        }
        offset *= 2;

    }

    if (thid==0) { temp[n -1] = 0;  }   // clear the last element

    for (int d =1; d < n; d *= 2)   // traverse down tree & build scan
    {
        offset >>= 1;
        __syncthreads();

        if (thid < d)
        {
            int ai = offset*(2*thid+1)-1;
            int bi = offset*(2*thid+2)-1;

            float t = temp[ai];
            temp[ai] = temp[bi];
            temp[bi] += t;
        }
    }

    __syncthreads();

    g_odata[2*thid] = temp[2*thid]; // write results to device memory
    g_odata[2*thid+1] = temp[2*thid+1];
}